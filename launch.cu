#include <stdio.h>
#include <hip/hip_runtime.h>
#include "op.h"
#include "kernels.h"
#include "tensor.h"

extern "C" void launchSoftmax(float *A, float *B, int J, int K) {
  Tensor<float> *t_A = new Tensor<float>({J, K}, A);
  Tensor<float> *t_B = new Tensor<float>({J, K}, B);

  std::vector<Tensor<float>*> tensors = {t_A, t_B};

  Softmax<float> op = Softmax<float>(tensors);

  op.forward();

  float* result = t_B->to_host();
  memcpy(B, result, t_B->n_bytes());

  free(result);

  delete t_A;
  delete t_B;
}

extern "C" void launchGradient(float *A, uint32_t *Y, float *B, int J, int K) {
  float *d_A, *d_B;
  uint32_t *d_Y;

  size_t sz_a = J * K * sizeof(float);
  size_t sz_y = J * sizeof(uint32_t);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_a);
  hipMalloc((void**)&d_Y, sz_y);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_Y, Y, sz_y, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE);
  dim3 gridDim(J);

  gradient<float> <<<gridDim, blockDim>>>(d_A, d_Y, d_B, J, K);

  hipMemcpy(B, d_B, sz_a, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_Y);
}

extern "C" void launchBiasAdd(float *A, float *B, float *C, int J, int K) {
  Tensor<float> *t_A = new Tensor<float>({J, K}, A);
  Tensor<float> *t_B = new Tensor<float>({K}, B);
  Tensor<float> *t_C = new Tensor<float>({J, K}, C);

  std::vector<Tensor<float>*> tensors = {t_A, t_B, t_C};

  BiasAdd<float> op = BiasAdd<float>(tensors);

  op.forward();

  float* result = t_C->to_host();

  memcpy(C, result, t_C->n_bytes());

  free(result);

  delete t_A;
  delete t_B;
  delete t_C;
}

extern "C" void launchScalarAdd(float *A, float *B, float S, int N) {
  float *d_A, *d_B;

  size_t sz_a = N * sizeof(float);
  size_t sz_b = N * sizeof(float);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE);
  dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

  scalarAdd<float> <<<gridDim, blockDim>>>(d_A, d_B, S, N);

  hipMemcpy(B, d_B, sz_b, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
}

extern "C" void launchRelu(float *A, float *B, int N) {
  Tensor<float> *t_A = new Tensor<float>({N}, A);
  Tensor<float> *t_B = new Tensor<float>({N}, B);

  std::vector<Tensor<float>*> tensors = {t_A, t_B};

  Relu<float> op = Relu<float>(tensors);
  op.forward();

  float* result = t_B->to_host();
  memcpy(B, result, t_B->n_bytes());
  free(result);

  delete t_A;
  delete t_B;
}

extern "C" void launchSumCols(float *A, float *B, int J, int K) {
  float *d_A, *d_B;

  size_t sz_a = J * K * sizeof(float);
  size_t sz_b = K * sizeof(float);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE);
  dim3 gridDim(K);

  sumCols<float> <<<gridDim, blockDim>>>(d_A, d_B, J, K);

  hipMemcpy(B, d_B, sz_b, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
}

extern "C" void launchMultInt(int *A, int *B, int *C, int J, int K, int M, int N) {

  int *d_A, *d_B, *d_C;

  size_t sz_a = J * K * sizeof(int);
  size_t sz_b = M * N * sizeof(int);
  size_t sz_c = J * N * sizeof(int);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);
  hipMalloc((void**)&d_C, sz_c);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (J + BLOCK_SIZE - 1) / BLOCK_SIZE);

  gemm<int><<<gridDim, blockDim>>>(d_A, d_B, d_C, J, K, M, N);

  hipMemcpy(C, d_C, sz_c, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

extern "C" void launchMult(float *A, float *B, float *C, int J, int K, int M, int N) {

  float *d_A, *d_B, *d_C;

  size_t sz_a = J * K * sizeof(float);
  size_t sz_b = M * N * sizeof(float);
  size_t sz_c = J * N * sizeof(float);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);
  hipMalloc((void**)&d_C, sz_c);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (J + BLOCK_SIZE - 1) / BLOCK_SIZE);

  gemm<float><<<gridDim, blockDim>>>(d_A, d_B, d_C, J, K, M, N);

  hipMemcpy(C, d_C, sz_c, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

extern "C" void launchTranspose2(float *A, float *B, float *C, int J, int K, int M, int N) {
  float *d_A, *d_B, *d_C;

  size_t sz_a = J * K * sizeof(float);
  size_t sz_b = M * N * sizeof(float);
  size_t sz_c = K * N * sizeof(float);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);
  hipMalloc((void**)&d_C, sz_c);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, sz_c, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (K + BLOCK_SIZE - 1) / BLOCK_SIZE);

  gemm2<true, false, float><<<gridDim, blockDim>>>(d_A, d_B, d_C, J, K, M, N);


  hipMemcpy(C, d_C, sz_c, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

extern "C" void launchTranspose(float *A, float *B, float *C, int J, int K, int M, int N) {
  float *d_A, *d_B, *d_C;

  size_t sz_a = J * K * sizeof(float);
  size_t sz_b = M * N * sizeof(float);
  size_t sz_c = J * M * sizeof(float);

  hipMalloc((void**)&d_A, sz_a);
  hipMalloc((void**)&d_B, sz_b);
  hipMalloc((void**)&d_C, sz_c);

  hipMemcpy(d_A, A, sz_a, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sz_b, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, sz_c, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((M + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (J + BLOCK_SIZE - 1) / BLOCK_SIZE);

  gemm2<false, true, float><<<gridDim, blockDim>>>(d_A, d_B, d_C, J, K, M, N);


  hipMemcpy(C, d_C, sz_c, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

extern "C" void launchMult2(float *A, float *B, float *C, int J, int K, int M, int N, bool backward) {
  Tensor<float> *t_A = new Tensor<float>({J, K}, A);
  Tensor<float> *t_B = new Tensor<float>({M, N}, B);

  Tensor<float> *t_C = new Tensor<float>({J, N}, C);
  std::vector<Tensor<float>*> tensors = {t_A, t_B, t_C};

  Gemm<float> op = Gemm<float>(tensors);

  op.forward();
  float *result1 = t_C->to_host();
  memcpy(C, result1, t_C->n_bytes());
  free(result1);

  if(backward) {
    op.backward();
    float *result = t_A->grad_to_host();
    memcpy(A, result, t_A->n_bytes());
    free(result);
    
    float *result2 = t_B->grad_to_host();
    memcpy(B, result2, t_B->n_bytes());
    free(result2);
  }


  delete t_A;
  delete t_B;
  delete t_C;
}

extern "C" void launchAdd(float* a, float* b, float* c, int J, int K) {
  Tensor<float> *t_a = new Tensor<float>({J, K}, a);
  Tensor<float> *t_b = new Tensor<float>({J, K}, b);
  Tensor<float> *t_c = new Tensor<float>({J, K}, c);

  std::vector<Tensor<float>*> tensors = {t_a, t_b, t_c};

  MatAdd<float> op = MatAdd<float>(tensors);

  op.forward();

  float* result = t_c->to_host();

  memcpy(c, result, t_c->n_bytes());

  free(result);
}